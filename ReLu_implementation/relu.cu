#include <iostream>
#include <hip/hip_runtime.h>

const int N = 64;
const int size = N * sizeof(float);

__global__ void relu_gpu(float *d_in, float *d_out, int N)
{
    int idx = threadIdx.x; // 每一个线程的索引
    if (idx < N)
    {
        d_out[idx] = d_in[idx] > 0 ? d_in[idx] : 0;
    }
}

int main()
{
    float *h_in = (float *)malloc(size); // 给host(cpu)上分配float数组的内存空间
    float *h_out = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_in[i] = i;
    }

    // 1.在GPU上分配内存
    float *d_in = nullptr;
    float *d_out = nullptr;
    hipMalloc(&d_in, size); // 分配的GPU内存的指针会存储在d_in中，但为了修改它，需要传递指针的指针
    hipMalloc(&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    relu_gpu<<<1, N>>>(d_in, d_out, N);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        std::cout << h_out[i] << std::endl;
    }

    hipFree(d_in);
    hipFree(d_out);

    free(h_in);
    free(h_out);
}